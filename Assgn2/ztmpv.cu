#include "hip/hip_runtime.h"
/*
  Program to compare performance of CPU and GPU on ZTMPV operation

  SGEMV  performs the below matrix-vector operations

    x := op(A)*x

  where x is an n element vector and  A is an n by n unit, or non-unit, upper or lower triangular matrix

  Link to ZTMPV: https://www.netlib.org/lapack/explore-html/dc/dc1/group__complex16__blas__level2_gaed33e3470ec372c730960b6038d1e037.html#gaed33e3470ec372c730960b6038d1e037

*/

#include <iostream>
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hipblas.h>

#define DIM_N 4 // Numbers of row and columns in matrix A
#define UPLO HIPBLAS_FILL_MODE_LOWER // Upper triangular or lower triangular, other option: HIPBLAS_FILL_MODE_UPPER
#define DIAG HIPBLAS_DIAG_NON_UNIT // Unit or Non unit diagonal, other option: HIPBLAS_DIAG_NON_UNIT
#define TRANSA HIPBLAS_OP_N // Operation to be performed,  HIPBLAS_OP_N => op(A) = A, HIPBLAS_OP_T => op(A) = A**T, HIPBLAS_OP_T => op(A) = A**H
#define INCX 1 //INCX specifies the increment for the elements of X

#define THREADS_PER_BLOCK 16 // Threads to spin per block in GPU
#define EPSILON 1e-2 // Precision for verifying actual and computed values

// Fill the vector x with random initial values
void init_vals(float *x, int N){
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 1234ULL);
  hiprandGenerateUniform(prng, x, N);
  hiprandDestroyGenerator(prng);
}

// Fill the matrix A with random initial values
void init_A(float *A){
  // Input matrix are assumed to be in column major order

  if(DIAG == HIPBLAS_DIAG_UNIT){
    for(int i=0; i<DIM_N; i++) A[i + i * DIM_N]= 1.0;
  }
  if(UPLO == HIPBLAS_FILL_MODE_LOWER){
    for(int i=0; i<DIM_N; i++){
      for(int j=i+1; j<DIM_N; j++) A[i + j*DIM_N]= 0.0;
    }
  }
  else{
    for(int i=0; i<DIM_N; i++){
      for(int j=0; j<i; j++) A[i + j*DIM_N]= 0.0;
    }
  }
}

void init_B(const float *A, float *B){
  // Input matrix are assumed to be in column major order

  if(UPLO == HIPBLAS_FILL_MODE_LOWER){
    for(int i=0; i<DIM_N; i++){
      for(int j=0; j<=i; j++) B[i -j + j*DIM_N]= A[i + j*DIM_N];
    }
  }
  else{
    for(int i=0; i<DIM_N; i++){
      for(int j=i; j<DIM_N; j++) B[DIM_N + i - j - 1 + j*DIM_N]= A[i + j*DIM_N];
    }
  }
}

// cuBLAS Level 3 routine call to perform ZTMPV operaiton
float cublas_ztmpv(const float *A, float *x){
  // Input matrix are assumed to be in column major order

  // Events to measure performance
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipEventRecord(start);
  hipblasStbmv(handle, UPLO, TRANSA, DIAG, DIM_N, DIM_N-1, A, DIM_N, x, INCX);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);

  hipblasDestroy(handle);
  return ms;
}

//GPU Kernel method to compute single element of the resultant vector
__global__ void gpu_kernel(const float *A, const float *x, float *res, int N, int op, int uplo){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<N) {
    float sum = 0;
    if(uplo == 1){
      for(int j=0;j<=i;j++){
        if(op == 1)
          sum += A[i + j * N] * x[j];
        else
          sum += A[i * N + j] * x[j];
      }
    }
    else{
      for(int j=i;j<N;j++){
        if(op == 1)
          sum += A[i + j * N] * x[j];
        else
          sum += A[i * N + j] * x[j];
      }
    }
    res[i] = sum;
  }
}

// Performing ZTMPV operation on GPU
float gpu_ztmpv(const float *A, const float *x, float *res){
  // Input matrix are assumed to be in column major order

  // Events to measure performance
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int op = (TRANSA == HIPBLAS_OP_N) ? 1 : 0;
  int uplo = (UPLO == HIPBLAS_FILL_MODE_LOWER) ? 1 : 0;

  hipEventRecord(start);

  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
  dim3 numBlocks( (DIM_N + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);

  gpu_kernel<<<numBlocks, threadsPerBlock>>>(A, x, res, DIM_N, op, uplo);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);

  return ms;
}

// Performing ZTMPV operation on CPU
float cpu_ztmpv(const float *A, const float *x, float *res){
  // Input matrix are assumed to be in column major order

  //Record start time
  std::clock_t cpu_start = std::clock();

  for (int i = 0; i < DIM_N; i++) {
    float sum = 0.0;
    if(UPLO == HIPBLAS_FILL_MODE_LOWER){
      for (int j = 0; j <= i; j++){
        if(TRANSA == HIPBLAS_OP_N)
          sum += A[i + j * DIM_N] * x[j];
        else
          sum += A[i * DIM_N + j] * x[j];
      }
    }
    else{
      for (int j = i; j < DIM_N; j++){
        if(TRANSA == HIPBLAS_OP_N)
          sum += A[i + j * DIM_N] * x[j];
        else
          sum += A[i * DIM_N + j] * x[j];
      }
    }
    res[i] = sum;
  }

  //Record end time
  std::clock_t cpu_end = std::clock();

  //return time elapsed in micro second
  long double cpu_ms = 1000.0 * (cpu_end-cpu_start) / CLOCKS_PER_SEC;
  return cpu_ms;
}

//Function to cross check computed vectors
int check(const float *A, const float *B, const float *C, int N){
  for(int i=0; i<N; i++){
    if(abs(B[i] - A[i]) > EPSILON || abs(C[i] - A[i]) > EPSILON){
      std::cout<<"Index : "<<i<<"  CPU : "<<A[i]<<"  GPU : "<<B[i]<<"  CuBLAS : "<<C[i]<<"\n";
      return 1;
    }
  }
  return 0;
}

// Function to print matrix stored in column major order
void print_mat(const float *M, int N){
  for(int i=0; i < N; i++) std::cout<<M[i]<<" ";
  std::cout<<std::endl;
}

int main(){
  // Declare device side vectors
  thrust::device_vector<float> d_A(DIM_N * DIM_N);
  thrust::device_vector<float> d_B(DIM_N * DIM_N);
  thrust::device_vector<float> d_x(DIM_N);
  thrust::device_vector<float> d_res(DIM_N);

  // Declare host side vectors
  thrust::host_vector<float> h_A(DIM_N * DIM_N);
  thrust::host_vector<float> h_B(DIM_N * DIM_N);
  thrust::host_vector<float> h_x(DIM_N);
  thrust::host_vector<float> h_res_gpu(DIM_N);
  thrust::host_vector<float> h_res_cpu(DIM_N);
  thrust::host_vector<float> h_res_cublas(DIM_N);

  // Initialize values
  init_vals(thrust::raw_pointer_cast(d_x.data()), DIM_N);
  init_vals(thrust::raw_pointer_cast(d_A.data()), DIM_N*DIM_N);

  // Copy device data to host
  h_A = d_A;
  h_x = d_x;

  init_A(thrust::raw_pointer_cast(h_A.data()));
  init_B(thrust::raw_pointer_cast(h_A.data()), thrust::raw_pointer_cast(h_B.data()));

  // Copy host data to device
  d_A = h_A;
  d_B = h_B;

  //Perform operation on the CPU
  float cpu_time = cpu_ztmpv( thrust::raw_pointer_cast(h_A.data()), thrust::raw_pointer_cast(h_x.data()), thrust::raw_pointer_cast(h_res_cpu.data()) );
  std::cout<<"Computation compeleted on CPU\n";

  //Perform operation on the GPU
  float gpu_time = gpu_ztmpv( thrust::raw_pointer_cast(d_A.data()), thrust::raw_pointer_cast(d_x.data()), thrust::raw_pointer_cast(d_res.data()) );
  std::cout<<"Computation compeleted on GPU using custom routine\n";

  // Perform operation on the GPU using cuBLAS routine
  float cublas_time = cublas_ztmpv(thrust::raw_pointer_cast(d_B.data()), thrust::raw_pointer_cast(d_x.data()) );
  std::cout<<"Computation compeleted on GPU using CuBLAS routine\n";

  // Copy result to host
  h_res_cublas = d_x;
  h_res_gpu = d_res;

  int status = check(thrust::raw_pointer_cast(h_res_cpu.data()), thrust::raw_pointer_cast(h_res_gpu.data()), thrust::raw_pointer_cast(h_res_cublas.data()), DIM_N);

  if(status == 0) std::cout<<"\nComputed vectors verified. No mismatch found.\n\n";
  else std::cout<<"\nComputed vectors not verified. Mismatch found.\n\n";

  //Print Result
  std::cout << "Input Data Shape \n"
            << "A : " << DIM_N <<" * " << DIM_N << "\n"
            << "x : " << DIM_N <<" * 1\n"
            << std::endl;
  //Print Result
  std::cout << "Perfermance \n"
            << "CPU Time : " << cpu_time << " ms\n"
            << "GPU Time : " << gpu_time << " ms\n"
            << "CuBLAS Time : " << cublas_time << " ms\n"
            << std::endl;
  return 0;
}
