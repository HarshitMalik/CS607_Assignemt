#include "hip/hip_runtime.h"
/*
  Program to compare performance of CPU and GPU on SGMEV operation

  SGEMV  performs the below matrix-vector operations

    y := alpha*op(A)*x + beta*y

 where alpha and beta are scalars, x and y are vectors and A is an m by n matrix.

 Link to SGMEV: https://www.netlib.org/lapack/explore-html/d6/d30/group__single__blas__level2_gafc92361b74c6d41c7e5afa0aa5d13ec9.html#gafc92361b74c6d41c7e5afa0aa5d13ec9

*/

#include <iostream>
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hipblas.h>

#define DIM_M 1024 // Numbers of row in matrix A
#define DIM_N 1024 // Numbers of columns in matrix A
#define DIM_X 1024 // Dimension of vector x, must be equal to number of columns in A or A**T accordingly
#define DIM_Y 1024 // Dimension of vector y, must be equal to number of rows in A or A**T accordingly
#define ALPHA 0.3 // value of alpha
#define BETA 0.7 // Value of beta
#define TRANSA HIPBLAS_OP_N // Operation to be performed,  HIPBLAS_OP_N => op(A) = A, HIPBLAS_OP_T => op(A) = A**T

#define THREADS_PER_BLOCK 32

// Fill the matrix/vectors with random initial values
void init_vals(float *in, int N){
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(prng, 1234ULL);
  hiprandGenerateUniform(prng, in, N);
  hiprandDestroyGenerator(prng);
}

// cuBLAS Level 3 routine call to perform SGMEV operaiton
float cublas_sgmev(const float *A, const float *x, float *y){
  // Input matrix are assumed to be in column major order

  // Events to measure performance
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int m = DIM_Y; // rows in op(A)
  int n = 1;   // columns in x
  int k = DIM_X; // columns in op(A)
  const float alpha = ALPHA;
  const float beta = BETA;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipEventRecord(start);
  hipblasSgemm(handle, TRANSA, HIPBLAS_OP_N, m, n, k, &alpha, A, DIM_M, x, DIM_X, &beta, y, DIM_Y);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);

  hipblasDestroy(handle);
  return ms;
}

__global__ void gpu_kernel(const float *A, const float *x, const float *y, float *res, int rows, int cols, int op, int alpha, int beta){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<rows) {
    float sum = 0;
    for(int j=0;j<cols;j++) {
      if(op == 1)
        sum += A[i + j * rows] * x[j];
      else
        sum += A[i * cols + j] * x[j];
    }
    res[i] = alpha*sum + beta*y[i];
  }
}

// Performing SGMEV operation on GPU
float gpu_sgmev(const float *A, const float *x, const float *y, float *res){
  // Input matrix are assumed to be in column major order

  // Events to measure performance
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int op = (TRANSA == HIPBLAS_OP_N) ? 1: 0;

  hipEventRecord(start);

  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
  dim3 numBlocks( (DIM_Y + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);

  gpu_kernel<<<numBlocks, threadsPerBlock>>>(A, x, y,res, DIM_Y, DIM_X, op, ALPHA, BETA);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);

  return ms;
}



// Performing SGMEV operation on CPU
float cpu_sgmev(const float *A, const float *x, float *y, float *res){
  // Input matrix are assumed to be in column major order

  //Record start time
  std::clock_t cpu_start = std::clock();

  //Perform operation
  for (int i = 0; i < DIM_Y; i++) {
    float sum = 0.0;
    for (int j = 0; j < DIM_X; j++){
      if(TRANSA == HIPBLAS_OP_N)
        sum += A[i + j * DIM_Y] * x[j];
      else
        sum += A[i * DIM_X + j] * x[j];
    }
    res[i] = ALPHA*sum + BETA*y[i];
  }

  //Record end time
  std::clock_t cpu_end = std::clock();

  //return time elapsed in micro second
  long double cpu_ms = 1000.0 * (cpu_end-cpu_start) / CLOCKS_PER_SEC;
  return cpu_ms;
}

// Function to print matrix in column major order
void print_mat(const float *M, int size){
  for(int i=0; i < size; i++) std::cout<<M[i]<<" ";
  std::cout<<std::endl;
}

int main(){
  // Declare device side vectors
  thrust::device_vector<float> d_A(DIM_M * DIM_N);
  thrust::device_vector<float> d_x(DIM_X);
  thrust::device_vector<float> d_y(DIM_Y);
  thrust::device_vector<float> d_res(DIM_Y);

  // Declare host side vectors
  thrust::host_vector<float> h_A(DIM_M * DIM_N);
  thrust::host_vector<float> h_x(DIM_X);
  thrust::host_vector<float> h_y(DIM_Y);
  thrust::host_vector<float> h_res_gpu(DIM_Y);
  thrust::host_vector<float> h_res_cpu(DIM_Y);
  thrust::host_vector<float> h_res_cuBLAS(DIM_Y);

  // Initialize values
  init_vals(thrust::raw_pointer_cast(d_A.data()), DIM_M * DIM_N);
  init_vals(thrust::raw_pointer_cast(d_x.data()), DIM_X);
  init_vals(thrust::raw_pointer_cast(d_y.data()), DIM_Y);

  // Copy device data to host
  h_A = d_A;
  h_x = d_x;
  h_y = d_y;

  //Perform operation on the CPU
  float cpu_time = cpu_sgmev( thrust::raw_pointer_cast(h_A.data()), thrust::raw_pointer_cast(h_x.data()), thrust::raw_pointer_cast(h_y.data()), thrust::raw_pointer_cast(h_res_cpu.data()) );

  //Perform operation on the GPU
  float gpu_time = gpu_sgmev( thrust::raw_pointer_cast(d_A.data()), thrust::raw_pointer_cast(d_x.data()), thrust::raw_pointer_cast(d_y.data()), thrust::raw_pointer_cast(d_res.data()) );

  // Perform operation on the GPU using cuBLAS routine
  float cublas_time = cublas_sgmev(thrust::raw_pointer_cast(d_A.data()), thrust::raw_pointer_cast(d_x.data()), thrust::raw_pointer_cast(d_y.data()) );

  // Copy result to host
  h_res_cuBLAS = d_y;
  h_res_gpu = d_res;

  //print_mat(thrust::raw_pointer_cast(h_res_cuBLAS.data()), DIM_Y);
  //print_mat(thrust::raw_pointer_cast(h_res_gpu.data()), DIM_Y);
  //print_mat(thrust::raw_pointer_cast(h_res_cpu.data()), DIM_Y);

  //Print Result
  std::cout << "TEST COMPLETED \n"
            << "CPU Time : " << cpu_time << " ms\n"
            << "GPU Time : " << gpu_time << " ms\n"
            << "CuBLAS Time : " << cublas_time << " ms"
            << std::endl;

  return 0;
}
